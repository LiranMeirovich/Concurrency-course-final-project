
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define BLOCK_DIM 1024 // number of threads in a block

/* Here we do an inclusive scan of 'array' in place.
   'size' is the number of elements in 'array'.
   it should be a power of 2.
 
   We assume that 'array' is in shared memory so that there is no need to 
   copy it to shared memory here.
    */

__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus

/*
   This kernel compares the two strings s1 and s2. Both strings are
   terminated with a null byte.
   The result is an integer:  0, if s1 and s2 are equal;
                              a negative value if s1 is less than s2;
                              a positive value if s1 is greater than s2
   The argument 'result' is used to "return" the result.
   The arguments n1, n2 indicate the number of characters in s1 and s2, respectively
    (including the null byte at the end).
             
   We assume that the number of threads in a block is >= max(n1,n2).  
*/
__global__ void my_strcmp(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
    //better use would be to malloc exact size with maxSize but this is easier to
    //implement
    __shared__ int flags[BLOCK_DIM];

    //finds the smaller arr (the rest of the string isn't important)
    int maxSize = n1 < n2 ? n1 : n2;
    //this is a trick to use the \0 also inside the str without changing the scan code too much
    //\0 will be considered as part of the string and will assist us in determining when does the
    //the 2 string end up diverting
    maxSize += 1;



    
    int tid = threadIdx.x;
    
    if (tid < n1 && tid < n2)
        flags[tid] = s1[tid] != s2[tid];
    else
        flags[tid] = 0; 

    __syncthreads();  // wait until all threads write to flags

    scan_plus(flags, maxSize);
    
    __syncthreads(); // wait until all threads complete write to flags

    //Ask Gadi what to do with this case
    // if (tid == 0 && (n1 == 0 || n2 == 0))//both strings is length 1
    //      *result = -1; 
    if (flags[tid] == 1 && tid < n1 && tid < n2)
         *result = s1[tid] - s2[tid]; // at most one thread will do this 

}


int main(int argc, char **argv) 
{

	char *dev_s1, *dev_s2;
    int *dev_result;
#if 0
    char s1[] = "supercalifragilisticexpialidocious";
    char s2[] = "supercalifragilisticexpialidocious";
#endif
    const char *s1, *s2; 

    if (argc == 3) {
        s1 = strdup(argv[1]);
        s2 = strdup(argv[2]);
    }
    else if (argc == 1) {
        /* read 2 strings from the standard input */
        if (scanf("%ms %ms", &s1, &s2) != 2) {
            fprintf(stderr, "invalid input\n");
            exit(1);
        }
    }
    else {
        fprintf(stderr, "usage: %s [<first string> <second string>]\n", argv[0]);
        exit(1);
    }

    int n1 = strlen(s1)+1; // null byte at the end is also counted
    int n2 = strlen(s2)+1;
           
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_s1, n1);
    hipMalloc((void**)&dev_s2, n2);
    hipMalloc((void**)&dev_result, sizeof(int));
    
    hipMemcpy(dev_s1, s1, n1, hipMemcpyHostToDevice);
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
 
    my_strcmp<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
 
    // copy the result back from the GPU to the CPU
    int result;
    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("result is %d\n", result);
		
	    
    // free memory on the GPU side
    hipFree(dev_s1);
    hipFree(dev_s2);
    hipFree(dev_result);
}
